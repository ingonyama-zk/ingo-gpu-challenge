#include <cstdint>
#include <hip/hip_runtime.h>
#include <stdexcept>


namespace ptx {

    __device__ __forceinline__ uint32_t add_cc(const uint32_t x, const uint32_t y) {
        uint32_t result;
        asm volatile("add.cc.u32 %0, %1, %2;" : "=r"(result) : "r"(x), "r"(y));
        return result;
    }
    
    __device__ __forceinline__ uint32_t addc(const uint32_t x, const uint32_t y) {
        uint32_t result;
        asm volatile("addc.u32 %0, %1, %2;" : "=r"(result) : "r"(x), "r"(y));
        return result;
    }
    
    __device__ __forceinline__ uint32_t addc_cc(const uint32_t x, const uint32_t y) {
        uint32_t result;
        asm volatile("addc.cc.u32 %0, %1, %2;" : "=r"(result) : "r"(x), "r"(y));
        return result;
    }

    __device__ __forceinline__ uint32_t sub_cc(const uint32_t x, const uint32_t y) {
        uint32_t result;
        asm volatile("sub.cc.u32 %0, %1, %2;" : "=r"(result) : "r"(x), "r"(y));
        return result;
      }
      
      __device__ __forceinline__ uint32_t subc(const uint32_t x, const uint32_t y) {
        uint32_t result;
        asm volatile("subc.u32 %0, %1, %2;" : "=r"(result) : "r"(x), "r"(y));
        return result;
      }
      
      __device__ __forceinline__ uint32_t subc_cc(const uint32_t x, const uint32_t y) {
        uint32_t result;
        asm volatile("subc.cc.u32 %0, %1, %2;" : "=r"(result) : "r"(x), "r"(y));
        return result;
      }      
    
    __device__ __forceinline__ uint32_t mul_lo(const uint32_t x, const uint32_t y) {
        uint32_t result;
        asm("mul.lo.u32 %0, %1, %2;" : "=r"(result) : "r"(x), "r"(y));
        return result;
    }
    
    __device__ __forceinline__ uint32_t mul_hi(const uint32_t x, const uint32_t y) {
        uint32_t result;
        asm("mul.hi.u32 %0, %1, %2;" : "=r"(result) : "r"(x), "r"(y));
        return result;
    }
    
    __device__ __forceinline__ uint32_t mad_lo_cc(const uint32_t x, const uint32_t y, const uint32_t z) {
        uint32_t result;
        asm volatile("mad.lo.cc.u32 %0, %1, %2, %3;" : "=r"(result) : "r"(x), "r"(y), "r"(z));
        return result;
    }

    __device__ __forceinline__ uint32_t mad_lo(const uint32_t x, const uint32_t y, const uint32_t z) {
        uint32_t result;
        asm volatile("mad.lo.u32 %0, %1, %2, %3;" : "=r"(result) : "r"(x), "r"(y), "r"(z));
        return result;
    }

    __device__ __forceinline__ uint32_t madc_lo(const uint32_t x, const uint32_t y, const uint32_t z) {
        uint32_t result;
        asm volatile("madc.lo.u32 %0, %1, %2, %3;" : "=r"(result) : "r"(x), "r"(y), "r"(z));
        return result;
    }

    __device__ __forceinline__ uint32_t madc_hi(const uint32_t x, const uint32_t y, const uint32_t z) {
        uint32_t result;
        asm volatile("madc.hi.u32 %0, %1, %2, %3;" : "=r"(result) : "r"(x), "r"(y), "r"(z));
        return result;
    }
    
    __device__ __forceinline__ uint32_t madc_lo_cc(const uint32_t x, const uint32_t y, const uint32_t z) {
        uint32_t result;
        asm volatile("madc.lo.cc.u32 %0, %1, %2, %3;" : "=r"(result) : "r"(x), "r"(y), "r"(z));
        return result;
    }
    
    __device__ __forceinline__ uint32_t mad_hi_cc(const uint32_t x, const uint32_t y, const uint32_t z) {
        uint32_t result;
        asm volatile("mad.hi.cc.u32 %0, %1, %2, %3;" : "=r"(result) : "r"(x), "r"(y), "r"(z));
        return result;
    }

    __device__ __forceinline__ uint32_t madc_hi_cc(const uint32_t x, const uint32_t y, const uint32_t z) {
        uint32_t result;
        asm volatile("madc.hi.cc.u32 %0, %1, %2, %3;" : "=r"(result) : "r"(x), "r"(y), "r"(z));
        return result;
    }

} // namespace ptx


// stands for "total limbs count"
const int TLC = 8;

struct __align__(16) bigint {
    uint32_t limbs[TLC];
};

struct __align__(16) bigint_wide {
    uint32_t limbs[2 * TLC];
};

static __device__ __forceinline__ void mul_n(uint32_t *acc, const uint32_t *a, uint32_t bi, size_t n = TLC) {
    size_t i;
    #pragma unroll
    for (i = 0; i < n - 2; i += 2) {
        acc[i] = ptx::mul_lo(a[i], bi);
        acc[i + 1] = ptx::mul_hi(a[i], bi);
    }
    acc[i] = ptx::mul_lo(a[i], bi);
    if (i == n - 2) acc[i + 1] = ptx::mul_hi(a[i], bi);
}

static __device__ __forceinline__ uint32_t mul_n_plus_extra(uint32_t *acc, const uint32_t *a, uint32_t bi, uint32_t *extra, size_t n = (TLC >> 1)) {
    acc[0] = ptx::mad_lo_cc(a[0], bi, extra[0]);

    #pragma unroll
    for (size_t i = 1; i < n - 1; i += 2) {
        acc[i] = ptx::madc_hi_cc(a[i - 1], bi, extra[i]);
        acc[i + 1] = ptx::madc_lo_cc(a[i + 1], bi, extra[i + 1]);
    }

    acc[n - 1] = ptx::madc_hi_cc(a[n - 2], bi, extra[n - 1]);
    return ptx::addc(0, 0);
}

template <bool CARRY_IN = false>
static __device__ __forceinline__ void cmad_n(uint32_t *acc, const uint32_t *a, uint32_t bi, size_t n = TLC, uint32_t optional_carry = 0) {
    if (CARRY_IN)
        ptx::add_cc(UINT32_MAX, optional_carry);
    acc[0] = CARRY_IN ? ptx::madc_lo_cc(a[0], bi, acc[0]) : ptx::mad_lo_cc(a[0], bi, acc[0]);
    acc[1] = ptx::madc_hi_cc(a[0], bi, acc[1]);

    #pragma unroll
    for (size_t i = 2; i < n; i += 2) {
        acc[i] = ptx::madc_lo_cc(a[i], bi, acc[i]);
        acc[i + 1] = ptx::madc_hi_cc(a[i], bi, acc[i + 1]);
    }
}

template <bool EVEN_PHASE>
static __device__ __forceinline__ void cmad_n_msb(uint32_t *acc, const uint32_t *a, uint32_t bi, size_t n = TLC) {
    if (EVEN_PHASE) {
        acc[0] = ptx::mad_lo_cc(a[0], bi, acc[0]);
        acc[1] = ptx::madc_hi_cc(a[0], bi, acc[1]);
    } else {
        acc[1] = ptx::mad_hi_cc(a[0], bi, acc[1]);
    }

    #pragma unroll
    for (size_t i = 2; i < n; i += 2) {
        acc[i] = ptx::madc_lo_cc(a[i], bi, acc[i]);
        acc[i + 1] = ptx::madc_hi_cc(a[i], bi, acc[i + 1]);
    }
}

template <bool CARRY_OUT = false, bool CARRY_IN = false>
static __device__ __forceinline__ uint32_t mad_row(uint32_t *odd, uint32_t *even, const uint32_t *a, uint32_t bi, size_t n = TLC, uint32_t ci = 0, uint32_t di = 0, uint32_t carry_for_high = 0, uint32_t carry_for_low = 0) {
    cmad_n<CARRY_IN>(odd, a + 1, bi, n - 2, carry_for_low);
    odd[n - 2] = ptx::madc_lo_cc(a[n - 1], bi, ci);
    odd[n - 1] = CARRY_OUT ? ptx::madc_hi_cc(a[n - 1], bi, di) : ptx::madc_hi(a[n - 1], bi, di);
    uint32_t cr = CARRY_OUT ? ptx::addc(0, 0) : 0;
    cmad_n(even, a, bi, n);
    odd[n - 1] = CARRY_OUT ? ptx::addc_cc(odd[n - 1], carry_for_high) : ptx::addc(odd[n - 1], carry_for_high);
    if (CARRY_OUT)
        cr = ptx::addc(cr, 0);
    return cr;
}

template <bool EVEN_PHASE>
static __device__ __forceinline__ void mad_row_msb(uint32_t *odd, uint32_t *even, const uint32_t *a, uint32_t bi, size_t n = TLC) {
    cmad_n_msb<!EVEN_PHASE>(odd, EVEN_PHASE ? a : (a + 1), bi, n - 2);
    odd[EVEN_PHASE ? (n - 1) : (n - 2)] = ptx::madc_lo_cc(a[n - 1], bi, 0);
    odd[EVEN_PHASE ? n : (n - 1)] = ptx::madc_hi(a[n - 1], bi, 0);
    cmad_n_msb<EVEN_PHASE>(even, EVEN_PHASE ? (a + 1) : a, bi, n - 1);
    odd[EVEN_PHASE ? n : (n - 1)] = ptx::addc(odd[EVEN_PHASE ? n : (n - 1)], 0);
}

static __device__ __forceinline__ void cmad_n_lsb(uint32_t *acc, const uint32_t *a, uint32_t bi, size_t n = TLC) {
    if (n > 1)
        acc[0] = ptx::mad_lo_cc(a[0], bi, acc[0]);
    else
        acc[0] = ptx::mad_lo(a[0], bi, acc[0]);

    size_t i;
    #pragma unroll
    for (i = 1; i < n - 1; i += 2) {
        acc[i] = ptx::madc_hi_cc(a[i - 1], bi, acc[i]);
        if (i == n - 2)
            acc[i + 1] = ptx::madc_lo(a[i + 1], bi, acc[i + 1]);
        else
            acc[i + 1] = ptx::madc_lo_cc(a[i + 1], bi, acc[i + 1]);
    }
    if (i == n - 1) acc[i] = ptx::madc_hi(a[i - 1], bi, acc[i]);
}

static __device__ __forceinline__ void mad_row_lsb(uint32_t *odd, uint32_t *even, const uint32_t *a, uint32_t bi, size_t n = TLC) {
    if (bi != 0) {
        if (n > 1) cmad_n_lsb(odd, a + 1, bi, n - 1);
        cmad_n_lsb(even, a, bi, n);
    }
    return;
}

template <bool SUBTRACT, bool CARRY_OUT> 
static constexpr __device__ __forceinline__ uint32_t add_sub_limbs_device(const uint32_t *x, const uint32_t *y, uint32_t *r, size_t n = (TLC >> 1)) {
    r[0] = SUBTRACT ? ptx::sub_cc(x[0], y[0]) : ptx::add_cc(x[0], y[0]);
    for (unsigned i = 1; i < (CARRY_OUT ? n : n - 1); i++)
        r[i] = SUBTRACT ? ptx::subc_cc(x[i], y[i]) : ptx::addc_cc(x[i], y[i]);
    if (!CARRY_OUT) {
        r[n - 1] = SUBTRACT ? ptx::subc(x[n - 1], y[n - 1]) : ptx::addc(x[n - 1], y[n - 1]);
        return 0;
    }
    return SUBTRACT ? ptx::subc(0, 0) : ptx::addc(0, 0);
}

// This method multiplies `a` and `b` and adds `in1` and `in2` to the result
// It is used to compute the "middle" part of Karatsuba: `a0 * b1 + b0 * a1`
// So under the assumption that the top bits of `a` and `b` are unset, we can ignore all the carries from here
static __device__ __forceinline__ void multiply_and_add_short_raw_device(const uint32_t *a, const uint32_t *b, uint32_t *even, uint32_t *in1, uint32_t *in2) {
    __align__(16) uint32_t odd[TLC - 2];
    uint32_t first_row_carry = mul_n_plus_extra(even, a, b[0], in1);
    uint32_t carry = mul_n_plus_extra(odd, a + 1, b[0], &in2[1]);

    size_t i;
    #pragma unroll
    for (i = 2; i < ((TLC >> 1) - 1); i += 2) {
        carry = mad_row<true, false>(&even[i], &odd[i - 2], a, b[i - 1], TLC >> 1, in1[(TLC >> 1) + i - 2], in1[(TLC >> 1) + i - 1], carry);
        carry = mad_row<true, false>(&odd[i], &even[i], a, b[i], TLC >> 1, in2[(TLC >> 1) + i - 1], in2[(TLC >> 1) + i], carry);
    }
    mad_row<false, true>(&even[TLC >> 1], &odd[(TLC >> 1) - 2], a, b[(TLC >> 1) - 1], TLC >> 1, in1[TLC - 2], in1[TLC - 1], carry, first_row_carry);
    // merge |even| and |odd| plus the parts of in2 we haven't added yet
    even[0] = ptx::add_cc(even[0], in2[0]);
    for (i = 0; i < (TLC - 2); i++)
        even[i + 1] = ptx::addc_cc(even[i + 1], odd[i]);
    even[i + 1] = ptx::addc(even[i + 1], in2[i + 1]);
}

static __device__ __forceinline__ void multiply_short_raw_device(const uint32_t *a, const uint32_t *b, uint32_t *even) {
    __align__(16) uint32_t odd[TLC - 2];
    mul_n(even, a, b[0], TLC >> 1);
    mul_n(odd, a + 1, b[0], TLC >> 1);
    mad_row(&even[2], &odd[0], a, b[1], TLC >> 1);

    size_t i;
#pragma unroll
    for (i = 2; i < ((TLC >> 1) - 1); i += 2) {
        mad_row(&odd[i], &even[i], a, b[i], TLC >> 1);
        mad_row(&even[i + 2], &odd[i], a, b[i + 1], TLC >> 1);
    }
    // merge |even| and |odd|
    even[1] = ptx::add_cc(even[1], odd[0]);
    for (i = 1; i < TLC - 2; i++)
        even[i + 1] = ptx::addc_cc(even[i + 1], odd[i]);
    even[i + 1] = ptx::addc(even[i + 1], 0);
}

static __device__ __forceinline__ void multiply_and_add_lsb_raw_device(const bigint &as, const bigint &bs, bigint &cs, bigint_wide &rs) {
    // r = a * b is correct for the first TLC digits
    const uint32_t *a = as.limbs;
    const uint32_t *b = bs.limbs;
    uint32_t *even = rs.limbs;
    __align__(16) uint32_t odd[2 * TLC - 2];
    size_t i;
    if (b[0] == UINT32_MAX) {
        add_sub_limbs_device<true, false>(cs.limbs, a, even, TLC);
        for (i = 0; i < TLC - 1; i += 1)
            odd[i] = a[i];
    } else {
        mul_n_plus_extra(even, a, b[0], cs.limbs, TLC);
        mul_n(odd, a + 1, b[0], TLC - 1);
    }
    mad_row_lsb(&even[2], &odd[0], a, b[1], TLC - 1);
#pragma unroll
    for (i = 2; i < TLC - 1; i += 2) {
        mad_row_lsb(&odd[i], &even[i], a, b[i], TLC - i);
        mad_row_lsb(&even[i + 2], &odd[i], a, b[i + 1], TLC - i - 1);
    }

    // merge |even| and |odd|
    even[1] = ptx::add_cc(even[1], odd[0]);
    for (i = 1; i < TLC + 1; i++)
        even[i + 1] = ptx::addc_cc(even[i + 1], odd[i]);
    even[i + 1] = ptx::addc(even[i + 1], 0);
}

static __device__ __forceinline__ void multiply_msb_raw_device(const bigint& as, const bigint& bs, bigint_wide& rs) {
    // r = a * b is almost correct for the last TLC + 1 digits
    const uint32_t *a = as.limbs;
    const uint32_t *b = bs.limbs;
    uint32_t *even = rs.limbs;
    __align__(16) uint32_t odd[2 * TLC - 2];

    even[TLC - 1] = ptx::mul_hi(a[TLC - 2], b[0]);
    odd[TLC - 2] = ptx::mul_lo(a[TLC - 1], b[0]);
    odd[TLC - 1] = ptx::mul_hi(a[TLC - 1], b[0]);
    size_t i;
#pragma unroll
    for (i = 2; i < TLC - 1; i += 2) {
        mad_row_msb<true>(&even[TLC - 2], &odd[TLC - 2], &a[TLC - i - 1], b[i - 1], i + 1);
        mad_row_msb<false>(&odd[TLC - 2], &even[TLC - 2], &a[TLC - i - 2], b[i], i + 2);
    }
    mad_row(&even[TLC], &odd[TLC - 2], a, b[TLC - 1]);

    // merge |even| and |odd|
    ptx::add_cc(even[TLC - 1], odd[TLC - 2]);
    for (i = TLC - 1; i < 2 * TLC - 2; i++)
        even[i + 1] = ptx::addc_cc(even[i + 1], odd[i]);
    even[i + 1] = ptx::addc(even[i + 1], 0);
}

static constexpr unsigned slack_bits = 3;

static constexpr __device__ __forceinline__ bigint get_higher_with_slack(const bigint_wide &xs) {
    bigint out{};
    for (unsigned i = 0; i < TLC; i++) {
        out.limbs[i] = __funnelshift_lc(xs.limbs[i + TLC - 1], xs.limbs[i + TLC], 2 * slack_bits);
    }
    return out;
}

static constexpr __device__ __forceinline__ bigint get_higher(const bigint_wide &xs) {
    bigint out{};
    for (unsigned i = 0; i < TLC; i++) {
        out.limbs[i] = xs.limbs[i + TLC];
    }
    return out;
}

static constexpr __device__ __forceinline__ bigint get_lower(const bigint_wide &xs) {
    bigint out{};
    for (unsigned i = 0; i < TLC; i++)
        out.limbs[i] = xs.limbs[i];
    return out;
}

static __device__ __forceinline__ void multiply_raw_device(const bigint &as, const bigint &bs, bigint_wide &rs) {
    const uint32_t *a = as.limbs;
    const uint32_t *b = bs.limbs;
    uint32_t *r = rs.limbs;
    multiply_short_raw_device(a, b, r);
    multiply_short_raw_device(&a[TLC >> 1], &b[TLC >> 1], &r[TLC]);
    __align__(16) uint32_t middle_part[TLC];
    __align__(16) uint32_t diffs[TLC];
    uint32_t carry1 = add_sub_limbs_device<true, true>(&a[TLC >> 1], a, diffs);
    uint32_t carry2 = add_sub_limbs_device<true, true>(b, &b[TLC >> 1], &diffs[TLC >> 1]);
    multiply_and_add_short_raw_device(diffs, &diffs[TLC >> 1], middle_part, r, &r[TLC]);
    if (carry1)
        add_sub_limbs_device<true, false>(&middle_part[TLC >> 1], &diffs[TLC >> 1], &middle_part[TLC >> 1]);
    if (carry2)
        add_sub_limbs_device<true, false>(&middle_part[TLC >> 1], diffs, &middle_part[TLC >> 1]);
    add_sub_limbs_device<false, true>(&r[TLC >> 1], middle_part, &r[TLC >> 1], TLC);

    for (size_t i = TLC + (TLC >> 1); i <  2 * TLC; i++)
        r[i] = ptx::addc_cc(r[i], 0);
}

static constexpr __device__ __forceinline__ bigint get_m() {
    return bigint {0x151e79ea, 0xf5204c21, 0x8d69e258, 0xfd0a180b, 0xfaa80548, 0xe4e51e49, 0xc40b2c9e, 0x36d9491e};
}

static constexpr __device__ __forceinline__ bigint get_neg_modulus() {
    return bigint {0xffffffff, 0xf5ee7fff, 0x2ffffffe, 0xa6558901, 0xa3c84ffe, 0x9f4bb2e1, 0x65d35aa9, 0xed549aa1};
}

static constexpr __device__ __forceinline__ bigint_wide get_modulus_wide() {
    return bigint_wide {
      0x00000001, 0x0a118000, 0xd0000001, 0x59aa76fe, 0x5c37b001, 0x60b44d1e, 0x9a2ca556, 0x12ab655e,
      0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000};
}

static constexpr __device__ __forceinline__ bigint_wide get_two_modulus_wide() {
    return bigint_wide { 
      0x00000002, 0x14230000, 0xa0000002, 0xb354edfd, 0xb86f6002, 0xc1689a3c, 0x34594aac, 0x2556cabd,
      0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000};
}

static __device__ __forceinline__ uint32_t sub_limbs_partial_device(const bigint_wide &as, const bigint_wide &bs, bigint_wide &rs, uint32_t num_limbs) {
    const uint32_t *a = as.limbs;
    const uint32_t *b = bs.limbs;
    uint32_t *r = rs.limbs;
    return add_sub_limbs_device<true, true>(a, b, r, num_limbs);
}

// a method that reduces modulo some prime number (currently - bls12-377 scalar field prime)
static __device__ __forceinline__ bigint reduce(const bigint_wide& xs) {
    bigint xs_hi = get_higher_with_slack(xs); // xy << slack_bits
    bigint_wide l = {};
    multiply_msb_raw_device(xs_hi, get_m(), l);      // MSB mult
    bigint l_hi = get_higher(l);
    bigint_wide r_wide = {};
    bigint xs_lo = get_lower(xs);
    multiply_and_add_lsb_raw_device(l_hi, get_neg_modulus(), xs_lo, r_wide); // LSB mult
    bigint_wide r_wide_reduced = {};
    // uint32_t carry = sub_limbs_partial_device(r_wide, get_two_modulus_wide(), r_wide_reduced, TLC);
    // if (carry == 0) // continue to reduce
    //     r_wide = r_wide_reduced;
    uint32_t carry = sub_limbs_partial_device(r_wide, get_modulus_wide(), r_wide_reduced, TLC);
    if (carry == 0) // continue to reduce
        r_wide = r_wide_reduced;

    bigint r = get_lower(r_wide);
    return r;
}


// The kernel that does element-wise multiplication of arrays in1 and in2 N times
template <int N>
__global__ void multVectorsKernel(bigint *in1, const bigint *in2, bigint *out, size_t n)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < n)
    {
        bigint i1 = in1[tid];
        const bigint i2 = in2[tid];
        bigint_wide o = {0};
        for (int i = 0; i < N - 1; i++) {
            multiply_raw_device(i1, i2, o);
            i1 = reduce(o);
        }
        multiply_raw_device(i1, i2, o);
        out[tid] = reduce(o);
    }
}

template <int N>
int mult_vectors(bigint in1[], const bigint in2[], bigint *out, size_t n)
{
    // Set the grid and block dimensions
    int threads_per_block = 128;
    int num_blocks = (n + threads_per_block - 1) / threads_per_block + 1;

    multVectorsKernel<N><<<num_blocks, threads_per_block>>>(in1, in2, out, n);

    return 0;
}


extern "C"
int multiply_test(bigint in1[], const bigint in2[], bigint *out, size_t n)
{
    try
    {
        mult_vectors<1>(in1, in2, out, n);
        return hipSuccess;
    }
    catch (const std::runtime_error &ex)
    {
        return -1;
    }
}

extern "C"
int multiply_bench(bigint in1[], const bigint in2[], bigint *out, size_t n)
{
    try
    {
        // for benchmarking, we need to give each thread a number of multiplication tasks that would ensure
        // that we're mostly measuring compute and not global memory accesses, which is why we do 500 multiplications here
        mult_vectors<500>(in1, in2, out, n);
        return hipSuccess;
    }
    catch (const std::runtime_error &ex)
    {
        return -1;
    }
}
