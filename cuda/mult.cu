#include "hip/hip_runtime.h"
#include <cstdint>
#include <hip/hip_runtime.h>
#include <stdexcept>


namespace ptx {

    __device__ __forceinline__ uint32_t add_cc(const uint32_t x, const uint32_t y) {
        uint32_t result;
        asm volatile("add.cc.u32 %0, %1, %2;" : "=r"(result) : "r"(x), "r"(y));
        return result;
    }
    
    __device__ __forceinline__ uint32_t addc(const uint32_t x, const uint32_t y) {
        uint32_t result;
        asm volatile("addc.u32 %0, %1, %2;" : "=r"(result) : "r"(x), "r"(y));
        return result;
    }
    
    __device__ __forceinline__ uint32_t addc_cc(const uint32_t x, const uint32_t y) {
        uint32_t result;
        asm volatile("addc.cc.u32 %0, %1, %2;" : "=r"(result) : "r"(x), "r"(y));
        return result;
    }

    __device__ __forceinline__ uint32_t sub_cc(const uint32_t x, const uint32_t y) {
        uint32_t result;
        asm volatile("sub.cc.u32 %0, %1, %2;" : "=r"(result) : "r"(x), "r"(y));
        return result;
      }
      
      __device__ __forceinline__ uint32_t subc(const uint32_t x, const uint32_t y) {
        uint32_t result;
        asm volatile("subc.u32 %0, %1, %2;" : "=r"(result) : "r"(x), "r"(y));
        return result;
      }
      
      __device__ __forceinline__ uint32_t subc_cc(const uint32_t x, const uint32_t y) {
        uint32_t result;
        asm volatile("subc.cc.u32 %0, %1, %2;" : "=r"(result) : "r"(x), "r"(y));
        return result;
      }      
    
    __device__ __forceinline__ uint32_t mul_lo(const uint32_t x, const uint32_t y) {
        uint32_t result;
        asm("mul.lo.u32 %0, %1, %2;" : "=r"(result) : "r"(x), "r"(y));
        return result;
    }
    
    __device__ __forceinline__ uint32_t mul_hi(const uint32_t x, const uint32_t y) {
        uint32_t result;
        asm("mul.hi.u32 %0, %1, %2;" : "=r"(result) : "r"(x), "r"(y));
        return result;
    }
    
    __device__ __forceinline__ uint32_t mad_lo_cc(const uint32_t x, const uint32_t y, const uint32_t z) {
        uint32_t result;
        asm volatile("mad.lo.cc.u32 %0, %1, %2, %3;" : "=r"(result) : "r"(x), "r"(y), "r"(z));
        return result;
    }

    __device__ __forceinline__ uint32_t madc_hi(const uint32_t x, const uint32_t y, const uint32_t z) {
        uint32_t result;
        asm volatile("madc.hi.u32 %0, %1, %2, %3;" : "=r"(result) : "r"(x), "r"(y), "r"(z));
        return result;
    }
    
    __device__ __forceinline__ uint32_t madc_lo_cc(const uint32_t x, const uint32_t y, const uint32_t z) {
        uint32_t result;
        asm volatile("madc.lo.cc.u32 %0, %1, %2, %3;" : "=r"(result) : "r"(x), "r"(y), "r"(z));
        return result;
    }
    
    __device__ __forceinline__ uint32_t madc_hi_cc(const uint32_t x, const uint32_t y, const uint32_t z) {
        uint32_t result;
        asm volatile("madc.hi.cc.u32 %0, %1, %2, %3;" : "=r"(result) : "r"(x), "r"(y), "r"(z));
        return result;
    }

    __device__ __forceinline__ uint32_t sub_cc(const uint32_t x, const uint32_t y) {
        uint32_t result;
        asm volatile("sub.cc.u32 %0, %1, %2;" : "=r"(result) : "r"(x), "r"(y));
        return result;
    }

    __device__ __forceinline__ uint32_t subc(const uint32_t x, const uint32_t y) {
        uint32_t result;
        asm volatile("subc.u32 %0, %1, %2;" : "=r"(result) : "r"(x), "r"(y));
        return result;
    }

    __device__ __forceinline__ uint32_t subc_cc(const uint32_t x, const uint32_t y) {
        uint32_t result;
        asm volatile("subc.cc.u32 %0, %1, %2;" : "=r"(result) : "r"(x), "r"(y));
        return result;
    }

} // namespace ptx


// stands for "total limbs count"
const int TLC = 8;

struct __align__(16) bigint {
    uint32_t limbs[TLC];
};

struct __align__(16) bigint_wide {
    uint32_t limbs[2 * TLC];
};

static __device__ __forceinline__ void mul_n(uint32_t *acc, const uint32_t *a, uint32_t bi, size_t n = TLC) {
    #pragma unroll
    for (size_t i = 0; i < n; i += 2) {
        acc[i] = ptx::mul_lo(a[i], bi);
        acc[i + 1] = ptx::mul_hi(a[i], bi);
    }
}

static __device__ __forceinline__ uint32_t mul_n_plus_extra(uint32_t *acc, const uint32_t *a, uint32_t bi, uint32_t *extra, size_t n = (TLC >> 1)) {
    acc[0] = ptx::mad_lo_cc(a[0], bi, extra[0]);

    #pragma unroll
    for (size_t i = 1; i < n - 1; i += 2) {
        acc[i] = ptx::madc_hi_cc(a[i - 1], bi, extra[i]);
        acc[i + 1] = ptx::madc_lo_cc(a[i + 1], bi, extra[i + 1]);
    }

    acc[n - 1] = ptx::madc_hi_cc(a[n - 2], bi, extra[n - 1]);
    return ptx::addc(0, 0);
}

template <bool CARRY_IN = false>
static __device__ __forceinline__ void cmad_n(uint32_t *acc, const uint32_t *a, uint32_t bi, size_t n = TLC, uint32_t optional_carry = 0) {
    if (CARRY_IN)
        ptx::add_cc(UINT32_MAX, optional_carry);
    acc[0] = CARRY_IN ? ptx::madc_lo_cc(a[0], bi, acc[0]) : ptx::mad_lo_cc(a[0], bi, acc[0]);
    acc[1] = ptx::madc_hi_cc(a[0], bi, acc[1]);

    #pragma unroll
    for (size_t i = 2; i < n; i += 2) {
        acc[i] = ptx::madc_lo_cc(a[i], bi, acc[i]);
        acc[i + 1] = ptx::madc_hi_cc(a[i], bi, acc[i + 1]);
    }
}

template <bool CARRY_OUT = false, bool CARRY_IN = false>
static __device__ __forceinline__ uint32_t mad_row(uint32_t *odd, uint32_t *even, const uint32_t *a, uint32_t bi, size_t n = TLC, uint32_t ci = 0, uint32_t di = 0, uint32_t carry_for_high = 0, uint32_t carry_for_low = 0) {
    cmad_n<CARRY_IN>(odd, a + 1, bi, n - 2, carry_for_low);
    odd[n - 2] = ptx::madc_lo_cc(a[n - 1], bi, ci);
    odd[n - 1] = ptx::madc_hi_cc(a[n - 1], bi, di);
    uint32_t cr = CARRY_OUT ? ptx::addc(0, 0) : 0;
    cmad_n(even, a, bi, n);
    odd[n - 1] = ptx::addc_cc(odd[n - 1], carry_for_high);
    if (CARRY_OUT)
        cr = ptx::addc(cr, 0);
    return cr;
}

template <bool SUBTRACT, bool CARRY_OUT> 
static constexpr __device__ __forceinline__ uint32_t add_sub_limbs_device(const uint32_t *x, const uint32_t *y, uint32_t *r, size_t n = (TLC >> 1)) {
    r[0] = SUBTRACT ? ptx::sub_cc(x[0], y[0]) : ptx::add_cc(x[0], y[0]);
    for (unsigned i = 1; i < (CARRY_OUT ? n : n - 1); i++)
        r[i] = SUBTRACT ? ptx::subc_cc(x[i], y[i]) : ptx::addc_cc(x[i], y[i]);
    if (!CARRY_OUT) {
        r[n - 1] = SUBTRACT ? ptx::subc(x[n - 1], y[n - 1]) : ptx::addc(x[n - 1], y[n - 1]);
        return 0;
    }
    return SUBTRACT ? ptx::subc(0, 0) : ptx::addc(0, 0);
}

// This method multiplies `a` and `b` and adds `in1` and `in2` to the result
// It is used to compute the "middle" part of Karatsuba: `a0 * b1 + b0 * a1`
// So under the assumption that the top bits of `a` and `b` are unset, we can ignore all the carries from here
static __device__ __forceinline__ void multiply_and_add_short_raw_device(const uint32_t *a, const uint32_t *b, uint32_t *even, uint32_t *in1, uint32_t *in2) {
    __align__(8) uint32_t odd[TLC - 2];
    uint32_t first_row_carry = mul_n_plus_extra(even, a, b[0], in1);
    uint32_t carry = mul_n_plus_extra(odd, a + 1, b[0], &in2[1]);

    size_t i;
    #pragma unroll
    for (i = 2; i < ((TLC >> 1) - 1); i += 2) {
        carry = mad_row<true, false>(&even[i], &odd[i - 2], a, b[i - 1], TLC >> 1, in1[(TLC >> 1) + i - 2], in1[(TLC >> 1) + i - 1], carry);
        carry = mad_row<true, false>(&odd[i], &even[i], a, b[i], TLC >> 1, in2[(TLC >> 1) + i - 1], in2[(TLC >> 1) + i], carry);
    }
    mad_row<false, true>(&even[TLC >> 1], &odd[(TLC >> 1) - 2], a, b[(TLC >> 1) - 1], TLC >> 1, in1[TLC - 2], in1[TLC - 1], carry, first_row_carry);
    // merge |even| and |odd| plus the parts of in2 we haven't added yet
    even[0] = ptx::add_cc(even[0], in2[0]);
    for (i = 0; i < (TLC - 2); i++)
        even[i + 1] = ptx::addc_cc(even[i + 1], odd[i]);
    even[i + 1] = ptx::addc(even[i + 1], in2[i + 1]);
}

static __device__ __forceinline__ void multiply_short_raw_device(const uint32_t *a, const uint32_t *b, uint32_t *even) {
    __align__(8) uint32_t odd[TLC - 2];
    mul_n(even, a, b[0], TLC >> 1);
    mul_n(odd, a + 1, b[0], TLC >> 1);
    mad_row(&even[2], &odd[0], a, b[1], TLC >> 1);

    size_t i;
#pragma unroll
    for (i = 2; i < ((TLC >> 1) - 1); i += 2) {
        mad_row(&odd[i], &even[i], a, b[i], TLC >> 1);
        mad_row(&even[i + 2], &odd[i], a, b[i + 1], TLC >> 1);
    }
    // merge |even| and |odd|
    even[1] = ptx::add_cc(even[1], odd[0]);
    for (i = 1; i < TLC - 2; i++)
        even[i + 1] = ptx::addc_cc(even[i + 1], odd[i]);
    even[i + 1] = ptx::addc(even[i + 1], 0);
}

static __device__ __forceinline__ void multiply_lsb_raw_device(const bigint &as, const bigint &bs, bigint_wide &rs) {
    // r = a * b is correcrt for the first TLC + 1 digits. (not computing from TLC + 1 to 2*TLC - 2).
    const uint32_t *a = as.limbs;
    const uint32_t *b = bs.limbs;
    uint32_t *even = rs.limbs;
    __align__(8) uint32_t odd[2 * TLC - 2];
    mul_n(even, a, b[0]);
    mul_n(odd, a + 1, b[0]);
    mad_row(&even[2], &odd[0], a, b[1]);
    size_t i;
#pragma unroll
    for (i = 2; i < TLC - 1; i += 2) {
        mad_row(&odd[i], &even[i], a, b[i], TLC - i + 2);
        mad_row(&even[i + 2], &odd[i], a, b[i + 1], TLC - i + 2);
    }

    // merge |even| and |odd|
    even[1] = ptx::add_cc(even[1], odd[0]);
    for (i = 1; i < TLC + 1; i++)
    even[i + 1] = ptx::addc_cc(even[i + 1], odd[i]);
    even[i + 1] = ptx::addc(even[i + 1], 0);
}

static constexpr unsigned slack_bits = 1;

static constexpr __device__ __forceinline__ bigint get_higher_with_slack(const bigint_wide &xs) {
    bigint out{};
    for (unsigned i = 0; i < TLC; i++) {
        out.limbs[i] = __funnelshift_lc(xs.limbs[i + TLC - 1], xs.limbs[i + TLC], slack_bits);
    }
    return out;
static __device__ __forceinline__ void multiply_raw_device(const bigint &as, const bigint &bs, bigint_wide &rs) {
    const uint32_t *a = as.limbs;
    const uint32_t *b = bs.limbs;
    uint32_t *r = rs.limbs;
    multiply_short_raw_device(a, b, r);
    multiply_short_raw_device(&a[TLC >> 1], &b[TLC >> 1], &r[TLC]);
    __align__(16) uint32_t middle_part[TLC];
    __align__(16) uint32_t diffs[TLC];
    uint32_t carry1 = add_sub_limbs_device<true, true>(&a[TLC >> 1], a, diffs);
    uint32_t carry2 = add_sub_limbs_device<true, true>(b, &b[TLC >> 1], &diffs[TLC >> 1]);
    multiply_and_add_short_raw_device(diffs, &diffs[TLC >> 1], middle_part, r, &r[TLC]);
    if (carry1)
        add_sub_limbs_device<true, false>(&middle_part[TLC >> 1], &diffs[TLC >> 1], &middle_part[TLC >> 1]);
    if (carry2)
        add_sub_limbs_device<true, false>(&middle_part[TLC >> 1], diffs, &middle_part[TLC >> 1]);
    add_sub_limbs_device<false, true>(&r[TLC >> 1], middle_part, &r[TLC >> 1], TLC);

    for (size_t i = TLC + (TLC >> 1); i <  2 * TLC; i++)
        r[i] = ptx::addc_cc(r[i], 0);
}

static constexpr __device__ __forceinline__ bigint get_lower(const bigint_wide &xs) {
    bigint out{};
    for (unsigned i = 0; i < TLC; i++)
        out.limbs[i] = xs.limbs[i];
    return out;
}

static constexpr __device__ __forceinline__ bigint get_m() {
    return bigint { 0x830358e4, 0x509cde80, 0x2f92eb5c, 0xd9410fad, 0xc1f823b4, 0xe2d772d, 0x7fb78ddf, 0x8d54253b };
}

static constexpr __device__ __forceinline__ bigint get_modulus() {
    return bigint { 0x00000001, 0xffffffff, 0xfffe5bfe, 0x53bda402, 0x09a1d805, 0x3339d808, 0x299d7d48, 0x73eda753 };
}

static constexpr __device__ __forceinline__ bigint_wide get_modulus_wide() {
    return bigint_wide { 0x00000001, 0xffffffff, 0xfffe5bfe, 0x53bda402, 0x09a1d805, 0x3339d808, 0x299d7d48, 0x73eda753,
                         0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000 };
}

static __device__ __forceinline__ uint32_t sub_limbs_partial_device(const bigint_wide &as, const bigint_wide &bs, bigint_wide &rs, uint32_t num_limbs) {
    const uint32_t *a = as.limbs;
    const uint32_t *b = bs.limbs;
    uint32_t *r = rs.limbs;
    r[0] = ptx::sub_cc(a[0], b[0]);
#pragma unroll
    for (unsigned i = 1; i < num_limbs; i++)
        r[i] = ptx::subc_cc(a[i], b[i]);
    return ptx::subc(0, 0);
}

// a method that reduces modulo some prime number (currently - bls12-381 scalar field prime)
static __device__ __forceinline__ bigint reduce(const bigint_wide& xs) {
    bigint xs_hi = get_higher_with_slack(xs); // xy << slack_bits
    bigint_wide l = {};
    multiply_raw_device(xs_hi, get_m(), l);      // MSB mult
    bigint l_hi = get_higher_with_slack(l);
    bigint_wide lp = {};
    multiply_lsb_raw_device(l_hi, get_modulus(), lp); // LSB mult
    bigint_wide r_wide = {};
    sub_limbs_partial_device(xs, lp, r_wide, 2 * TLC); 
    bigint_wide r_wide_reduced = {};
    for (unsigned i = 0; i < 2; i++)
    {
        uint32_t carry = sub_limbs_partial_device(r_wide, get_modulus_wide(), r_wide_reduced, TLC + 1);
        if (carry == 0) // continue to reduce
            r_wide = r_wide_reduced;
        else // done
            break;
    }
    
    // number of wrap around is bounded by TLC +  1 times.
    bigint r = get_lower(r_wide);
    return r;
}


// The kernel that does element-wise multiplication of arrays in1 and in2 N times
template <int N>
__global__ void multVectorsKernel(bigint *in1, const bigint *in2, bigint *out, size_t n)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < n)
    {
        bigint i1 = in1[tid];
        const bigint i2 = in2[tid];
        bigint_wide o = {0};
        // #pragma unroll
        for (int i = 0; i < N - 1; i++) {
            multiply_raw_device(i1, i2, o);
            i1 = reduce(o);
        }
        multiply_raw_device(i1, i2, o);
        out[tid] = reduce(o);
    }
}

template <int N>
int mult_vectors(bigint in1[], const bigint in2[], bigint *out, size_t n)
{
    // Set the grid and block dimensions
    int threads_per_block = 128;
    int num_blocks = (n + threads_per_block - 1) / threads_per_block + 1;

    multVectorsKernel<N><<<num_blocks, threads_per_block>>>(in1, in2, out, n);

    return 0;
}


extern "C"
int multiply_test(bigint in1[], const bigint in2[], bigint *out, size_t n)
{
    try
    {
        mult_vectors<1>(in1, in2, out, n);
        return hipSuccess;
    }
    catch (const std::runtime_error &ex)
    {
        return -1;
    }
}

extern "C"
int multiply_bench(bigint in1[], const bigint in2[], bigint *out, size_t n)
{
    try
    {
        // for benchmarking, we need to give each thread a number of multiplication tasks that would ensure
        // that we're mostly measuring compute and not global memory accesses, which is why we do 500 multiplications here
        mult_vectors<500>(in1, in2, out, n);
        return hipSuccess;
    }
    catch (const std::runtime_error &ex)
    {
        return -1;
    }
}
